/* CUDA installation test
*/
/*
  Copyright (C) 2013  Xi'an Jiaotong University
	Author(s): Pengliang Yang

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <rsf.h>
}

static void sf_check_gpu_error (const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) 
        sf_error ("Cuda error: %s: %s", msg, hipGetErrorString (err));
}


int main(int argc, char *argv[])
{

    hipSetDevice (0);
    sf_check_gpu_error ("Device initialization");

	printf("success!\n");
	
	exit(0);
}
