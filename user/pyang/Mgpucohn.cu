#include "hip/hip_runtime.h"
/* 3D coherence computation using GPU	
*/

/*
  Copyright (C) 2014 Xi'an Jiaotong University, UT Austin (Pengliang Yang)
   
  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.
   
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
   
  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

  Reference: Marfurt, Kurt J., et al. "Coherency calculations in the presence 
	of structural dip." Geophysics 64.1 (1999): 104-111.

*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

extern "C" {
#include <rsf.h>
}

#define BlockSizeX 16
#define BlockSizeY 16

static const int ntw=10;// window radius in t
static const int nxw=1;// window radius in x
static const int nyw=1;// window radius in y


static float *d_u1, *d_u2;

__global__ void coh3(float *u1, float *u2, int dimx, int dimy, int dimz)
/*< C3 calculation
NB: kernel configuration <<<gridDim, blockDim, sizeofsharedmembite>>>  >*/
{
	const int ix=threadIdx.x+blockIdx.x*blockDim.x;
	const int iy=threadIdx.y+blockIdx.y*blockDim.y;
	int id=ix+iy*dimx;

	__shared__ float s_u[BlockSizeY+2*nyw][BlockSizeX+2*nxw];
	const int tx=threadIdx.x+nxw;
	const int ty=threadIdx.y+nyw;

	const int J=(2*nxw+1)*(2*nyw+1);
	const int stride=dimx*dimy;

	float cxy[J][J];
	float u[J];
	float v[J];
	for(int iz=0; iz<dimz; iz++)
	{

		for(int i2=0; i2<J; i2++)
		for(int i1=0; i1<J; i1++)
			cxy[i2][i1]=0.0;
		for(int izw=-ntw; izw<=ntw; izw++)
		if(iz+izw>=0 && iz+izw<dimz)
		{
			int idtmp=id+izw*stride;
			if(threadIdx.y<nyw)// halo above/below
			{
				s_u[threadIdx.y][tx]=(blockIdx.y)?u1[idtmp-nyw*dimx]:0.0;
				s_u[threadIdx.y+BlockSizeY+nyw][tx]=(blockIdx.y<blockDim.y-1)?u1[idtmp+BlockSizeY*dimx]:0.0;
			}
			if(threadIdx.x<nxw)// halo left/right
			{
				s_u[tx][threadIdx.x]=(blockIdx.x)?u1[idtmp-nxw]:0.0;
				s_u[tx][threadIdx.x+BlockSizeX+nxw]=(blockIdx.x<blockDim.x-1)?u1[idtmp+BlockSizeX]:0.0;
			}
		
			for(int iy1=-nyw; iy1<=nyw; iy1++)
			for(int ix1=-nxw; ix1<=nxw; ix1++)
			for(int iy2=-nyw; iy2<=nyw; iy2++)
			for(int ix2=-nxw; ix2<=nxw; ix2++)
			{
				int ixc1=ix+ix1;
				int iyc1=iy+iy1;
				int ixc2=ix+ix2;
				int iyc2=iy+iy2;
				// check in bounds
				if( (ixc1>=0)&&(ixc1<dimx)&&
				    (iyc1>=0)&&(iyc2<dimy)&&
				    (ixc2>=0)&&(ixc2<dimx)&&
				    (iyc2>=0)&&(iyc2<dimy))
				cxy[ix2+nxw+(2*nxw+1)*(iy2+nyw)][ix1+nxw+(2*nxw+1)*(iy1+nyw)]+=s_u[iy1+ty][ix1+tx]*s_u[iy2+ty][ix2+tx];
			}	
		}
		/************************ C3 calculation  ********************/
		float s, m1, m;
		int i,j,k, maxidx;

		s=m1=0;
		for(i=0; i<J; i++) 
		{
			s+=cxy[i][i];// trace{cxy}
			u[i]=1.0;//initialize u
		}

		for(k=0; k<30; k++){//iterations
			for(i=0; i<J; i++){
				v[i]=0.0;
				for(j=0; j<J; j++)
				v[i]+=cxy[i][j]*u[j];
			}

			m=fabsf(v[0]);	maxidx=0;
			for(i=0; i<J; i++){
				 maxidx=(m>fabsf(v[i]))?maxidx:i;
				 m=(m>fabsf(v[i]))?m:fabsf(v[i]);
			}
			m=v[maxidx];
			for(i=0; i<J; i++) u[i]=v[i]/m;

			if(fabsf(m-m1)<1.e-6) break;
			m1=m;
		}

		/************************ End C3 calculation *****************/
		id+=stride;
	}
	

}

int main(int argc, char *argv[])
{
    	sf_file in, out;
    	int n1, n2, n3;
	float ***u1, ***u2;

    	sf_init(argc, argv);
    	in=sf_input("in");	/* 3D seismic data volume */
   	out=sf_output("out");	/* 3D coherence volume */

    	if (!sf_histint(in,"n1",&n1)) 	sf_error("No n1= in input");
    	if (!sf_histint(in,"n2",&n2)) 	sf_error("No n2= in input");
    	if (!sf_histint(in,"n3",&n3)) 	n3=1;	/* default: n3=1 if 2D */

	u1 = sf_floatalloc3(n1, n2, n3);
	u2 = sf_floatalloc3(n1, n2, n3);
	sf_floatread(u1[0][0], n1*n2*n3, in);
	memset(u2[0][0],0, n1*n2*n3*sizeof(float));

    	hipSetDevice(0);
    	hipError_t err = hipGetLastError ();
    	if (hipSuccess != err) 
	sf_warning("Cuda error: Failed to initialize device: %s", hipGetErrorString(err));

	hipMalloc(&d_u1, n1*n2*n3*sizeof(float));
	hipMalloc(&d_u2, n1*n2*n3*sizeof(float));



    	hipMemcpy(d_u1, u1[0][0], n1*n2*n3*sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_u2, 0, n1*n2*n3*sizeof(float));




    	hipMemcpy(u2, d_u2, n1*n2*n3*sizeof(float), hipMemcpyDeviceToHost);
	sf_floatwrite(u2[0][0], n1*n2*n3, out);

	hipFree(d_u1);
	hipFree(d_u2);

	free(**u1); free(*u1); free(u1);
	free(**u2); free(*u2); free(u2);

    	exit(0);
}

